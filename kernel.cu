//C or C++ headers
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include<iostream>
#include<cmath>

//CUDA headers
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include ""
////CUDA thrust
//#include <thrust/host_vector.h>
//#include <thrust/device_vector.h>
//#include <thrust/copy.h>
//#include <thrust/for_each.h>

#include"cublas_wrapper.h"

int main(int argc, char * argv[])
{

	Deng::CUDA_Vec::Col a(10);
	Deng::CUDA_Vec::Col b(10);

	auto c = -b;

	return EXIT_SUCCESS;
}